#include "hip/hip_runtime.h"

#define BLOCK_SIZE 32

#define BUF_SZ 5

typedef unsigned char uchar;

__global__ void cuDiff(const float *dataset, float *result_image, int w, int h)
{
	int tx = threadIdx.x;   int ty = threadIdx.y;
	int bx = blockIdx.x;	int by = blockIdx.y;

	int gx = bx * BLOCK_SIZE + tx;
	int gy = by * BLOCK_SIZE + ty;
	int s_idx = gy * w + gx;
	
	float sum = 0;
#pragma unroll
	for (int n = 0; n < BUF_SZ; n++) {
		sum += dataset[s_idx * BUF_SZ + n];
	}
	float avg = sum / BUF_SZ;

	float sum_diff_sq = 0;

	for (int n = 0; n < BUF_SZ; n++) {
		float diff = dataset[s_idx * BUF_SZ + n] - avg;
		sum_diff_sq += diff * diff;
	}

	float sdev = sqrtf(sum_diff_sq / BUF_SZ);

	result_image[s_idx] = fabsf(avg - dataset[s_idx * BUF_SZ + BUF_SZ-1]) > sdev ? 255.0f : 0.0f;
}

extern "C" bool cuImageProcessing(uchar *dataset, uchar *res, int w, int h)
{
	// convert to float
	float *pinned_dataset, *pinned_result_image;
	float *dev_dataset, *dev_res;

	hipHostAlloc<float>((float**)&pinned_dataset, w * h * BUF_SZ * sizeof(float), hipHostMallocDefault);
	hipHostAlloc<float>((float**)&pinned_result_image, w * h * sizeof(float), hipHostMallocDefault);

	// replace inner pixels with image data
	for (int y = 0; y < h; y++)
		for (int x = 0; x < w; x++)
			for (int n = 0; n < BUF_SZ; n++)
				pinned_dataset[(y * w + x) * BUF_SZ + n] = (float) dataset[(y * w + x) * BUF_SZ + n];

	hipMalloc((void**)&dev_dataset, w * h * BUF_SZ * sizeof(float));
	hipMalloc((void**)&dev_res, w * h * sizeof(float));

	hipMemcpy(dev_dataset, pinned_dataset, w * h * BUF_SZ * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimGrid(w / BLOCK_SIZE, h / BLOCK_SIZE);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	cuDiff<<<dimGrid, dimBlock>>>(dev_dataset, dev_res, w, h);
	hipDeviceSynchronize();

	hipMemcpy(pinned_result_image, dev_res, w * h * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < w * h; i++) {
		res[i] = (uchar)pinned_result_image[i];
	}

	hipFree(dev_dataset);
	hipFree(dev_res);
	hipHostFree(pinned_dataset);
	hipHostFree(pinned_result_image);

	return true;
}